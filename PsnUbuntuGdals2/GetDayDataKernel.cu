#include "hip/hip_runtime.h"
/*The getDayData model*/
#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#include "StructHeader.h"

using namespace std;

__global__ void accum(const double *GPP, double *sum)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    sum[i] = sum[i] + GPP[i];
}

__global__ void GetMean(const double *SumGpp, const int *DaySize, double *MeanGpp)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    MeanGpp[i] = SumGpp[i] / DaySize[0];
}

__global__ void GetDayData(const double *DayGpp, const double *Lai, double *GetDayGpp)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    /// unit:umol co2/m2/day
    GetDayGpp[i] = DayGpp[i] * 24.0 * 3600.0;
    /// unit: kgC/m2/day
    GetDayGpp[i] = GetDayGpp[i] * 12.011 * pow(10, -9);
}

extern "C" void accum_C(int blocksPerGrid, int threadsPerBlock, const double *GPP, double *sum)
{
    accum<<<blocksPerGrid, threadsPerBlock>>>(GPP, sum);
}

extern "C" void GetMean_C(int blocksPerGrid, int threadsPerBlock, const double *SumGpp, const int *DaySize, double *MeanGpp)
{
    GetMean<<<blocksPerGrid, threadsPerBlock>>>(SumGpp, DaySize, MeanGpp);
}

extern "C" void GetDayData_C(int blocksPerGrid, int threadsPerBlock, const double *DayGpp, const double *Lai, double *GetDayGpp)
{
    GetDayData<<<blocksPerGrid, threadsPerBlock>>>(DayGpp, Lai, GetDayGpp);
}