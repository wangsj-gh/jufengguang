#include "hip/hip_runtime.h"
__global__ void parallel_reduce_kernel(float *d_out, float* d_in){
    int myID = threadIdx.x + blockIdx.x * blockDim.x;
    int tid = threadIdx.x;

    //divide threads into two parts according to threadID, and add the right part to the left one, lead to reducing half elements, called an iteration; iterate until left only one element
    for(unsigned int s = blockDim.x / 2 ; s>0; s>>=1){
        if(tid<s){
            d_in[myID] += d_in[myID + s];
        }
        __syncthreads(); //ensure all adds at one iteration are done
    }
    if (tid == 0){
        d_out[blockIdx.x] = d_in[myId];
    }
}

__global__ void parallel_shared_reduce_kernel(float *d_out, float* d_in){
    int myID = threadIdx.x + blockIdx.x * blockDim.x;
    int tid = threadIdx.x;
    extern __shared__ float sdata[];
    sdata[tid] = d_in[myID];
    __syncthreads();

    //divide threads into two parts according to threadID, and add the right part to the left one, lead to reducing half elements, called an iteration; iterate until left only one element
    for(unsigned int s = blockDim.x / 2 ; s>0; s>>=1){
        if(tid<s){
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads(); //ensure all adds at one iteration are done
    }
    if (tid == 0){
        d_out[blockIdx.x] = sdata[myId];
    }
}